#include "hip/hip_runtime.h"
extern "C" __global__ void addForces(const real* __restrict__ grads,
                                     long long* __restrict__ forceBuffers,
                                     int* __restrict__ atomIndex,
                                     int gradSign,
                                     int numAtoms,
                                     int paddedNumAtoms) {
  for (int atom = blockIdx.x * blockDim.x + threadIdx.x; atom < numAtoms; atom += blockDim.x * gridDim.x) {
    int index = atomIndex[atom];
    forceBuffers[atom + 0 * paddedNumAtoms] -= gradSign * (long long)(grads[3 * index + 0] * 0x100000000);
    forceBuffers[atom + 1 * paddedNumAtoms] -= gradSign * (long long)(grads[3 * index + 1] * 0x100000000);
    forceBuffers[atom + 2 * paddedNumAtoms] -= gradSign * (long long)(grads[3 * index + 2] * 0x100000000);
  }
}

extern "C" __global__ void copyInputs(real* __restrict__ posTensor,
                                      real* __restrict__ boxTensor,
                                      const real4* __restrict__ posq,
                                      int* __restrict__ atomIndex,
                                      int numAtoms,
                                      real4 periodicBoxVecX,
                                      real4 periodicBoxVecY,
                                      real4 periodicBoxVecZ) {
  for (int atom = blockIdx.x * blockDim.x + threadIdx.x; atom < numAtoms; atom += blockDim.x * gridDim.x) {
    real4 pos = posq[atom];
    int index = atomIndex[atom];
    posTensor[3 * index + 0] = pos.x;
    posTensor[3 * index + 1] = pos.y;
    posTensor[3 * index + 2] = pos.z;
  }
  if (blockIdx.x == 0 and threadIdx.x == 0) {
    boxTensor[0] = periodicBoxVecX.x;
    boxTensor[1] = periodicBoxVecX.y;
    boxTensor[2] = periodicBoxVecX.z;
    boxTensor[3] = periodicBoxVecY.x;
    boxTensor[4] = periodicBoxVecY.y;
    boxTensor[5] = periodicBoxVecY.z;
    boxTensor[6] = periodicBoxVecZ.x;
    boxTensor[7] = periodicBoxVecZ.y;
    boxTensor[8] = periodicBoxVecZ.z;
  }
}
